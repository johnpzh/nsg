//
// Created by Zhen Peng on 10/22/19.
//

//#include <efanna2e/index_nsg.h>
//#include <efanna2e/util.h>
//#include <chrono>
//#include <string>
//// Added by Johnpzh
//#include <omp.h>
//#include  "extras/utils.h"
#include <stdio.h>
#include <iostream>
#include "../include/index_nd_nsg.h"
#include "../include/index_cuda.cuh"
//// Ended by Johnpzh

void load_data(char *filename, float *&data, unsigned &num,
               unsigned &dim)
{  // load data with sift10K pattern
    std::ifstream in(filename, std::ios::binary);
    if (!in.is_open()) {
        fprintf(stderr, "Error: cannot open file %s\n", filename);
        exit(EXIT_FAILURE);
    }
    in.read((char *) &dim, 4);
    // std::cout<<"data dimension: "<<dim<<std::endl;
    in.seekg(0, std::ios::end);
    std::ios::pos_type ss = in.tellg();
    size_t fsize = (size_t) ss;
    num = (unsigned) (fsize / (dim + 1) / 4);
    data = new float[(size_t) num * (size_t) dim];

    in.seekg(0, std::ios::beg);
    for (size_t i = 0; i < num; i++) {
        in.seekg(4, std::ios::cur);
        in.read((char *) (data + i * dim), dim * 4);
    }
    in.close();
}

int main(int argc, char **argv)
{

    if (argc != 8) {
        std::cout << argv[0]
                  << " data_file query_file nsg_path search_L search_K result_path query_num_max"
                  << std::endl;
        exit(-1);
    }
    setbuf(stdout, nullptr); // Added by Johnpzh
    float *data_load = NULL;
    unsigned points_num, dim;
    load_data(argv[1], data_load, points_num, dim);
    float *query_load = NULL;
    unsigned query_num, query_dim;
    load_data(argv[2], query_load, query_num, query_dim);
    assert(dim == query_dim);
    // Added by Johnpzh
    {
        unsigned query_num_max = strtoull(argv[7], nullptr, 0);
        if (query_num > query_num_max) query_num = query_num_max;
//        printf("query_num: %u\n", query_num);
    }
    // Ended by Johnpzh

    unsigned L = (unsigned) atoi(argv[4]);
    unsigned K = (unsigned) atoi(argv[5]);

    if (L < K) {
        std::cout << "search_L cannot be smaller than search_K!" << std::endl;
        exit(-1);
    }

    // data_load = efanna2e::data_align(data_load, points_num, dim);//one must
    // align the data before build query_load = efanna2e::data_align(query_load,
    // query_num, query_dim);
    efanna2e::IndexNDNSG index(dim, points_num, nullptr);
    index.Load(argv[3]);
    index.OptimizeGraph(data_load);

    bool *d_switch;
    hipMalloc(&d_switch, sizeof(bool));
    efanna2e::cudaPrint<<<2, 2>>>(d_switch);
    bool h_switch;
    hipMemcpy(&h_switch, d_switch, sizeof(bool), hipMemcpyDeviceToHost);
    hipFree(d_switch);
    if (h_switch) {
        printf("Switched.\n");
    }

    return 0;
}
