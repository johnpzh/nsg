//
// Created by Zhen Peng on 10/17/19.
//

//#include "efanna2e/index_nsg.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
namespace efanna2e {

//__global__ void IndexNSG::SearchWithOptGraphOnCUDA(
//        bool *d_switch)
//{
//    int i = threadIdx.x;
//    int j = threadIdx.y;
//    int z = threadIdx.z;
//
//    *switch = true;
//
//    printf("i: %d j: %d z: %d\n", i, j, z);
//}
__global__ void cudaPrint(bool *d_switch)
{
    int i = threadIdx.x;
    int j = threadIdx.y;
    int z = threadIdx.z;

    *d_switch = true;

    printf("i: %d j: %d z: %d\n", i, j, z);
}
}