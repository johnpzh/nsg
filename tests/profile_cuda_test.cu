//
// Created by Zhen Peng on 10/22/19.
//

//#include <efanna2e/index_nsg.h>
//#include <efanna2e/util.h>
//#include <chrono>
//#include <string>
//// Added by Johnpzh
//#include <omp.h>
//#include  "extras/utils.h"
#include <stdio.h>
#include "extras/index_cuda.cuh"
//// Ended by Johnpzh


int main(int argc, char **argv)
{
    bool *d_switch;
    hipMalloc(&d_switch, sizeof(bool));
    efanna2e::cudaPrint<<<1, 1>>>(d_switch);
    bool h_switch;
    hipMemcpy(&h_switch, d_switch, sizeof(bool), hipMemcpyDeviceToHost);
    hipFree(d_switch);
    if (h_switch) {
        printf("Switched.\n");
    }

    return 0;
}
